
#include <hip/hip_runtime.h>
#include <iostream>
#define M 50
#define tpb 256
#define bpg 1

__device__
bool is_same_block(int element_i, int other_i) {
  return ((element_i + 10) / 10 == (other_i + 10) / 10);
}

__global__
void modify(int n, int *vector) {
  __shared__ int s[M];

  int i = threadIdx.x;
  if (i < n) {
    s[i] = i * 10 + 123 - 456;

    __syncthreads();
    if (i + 1 < n) {
      int val = s[(i + 1)];
      vector[i] = val;
    } else {
      vector[i] = 0;
    }
  }
}

int main() {
  int *vector, *d_vector;

  vector = new int[M];
  for (int i = 0; i < M; i += 1) {
    vector[i] = 0;
  }

  hipMalloc(&d_vector, M * sizeof(int));
  hipMemcpy(d_vector, vector, M * sizeof(int), hipMemcpyHostToDevice);

  modify<<<bpg, tpb>>>(M, d_vector);

  hipMemcpy(vector, d_vector, M * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < M; i += 1) {
    printf("%d\t", vector[i]);
  }

  hipFree(d_vector);
  free(vector);

  hipDeviceReset();

  return 0;
}
