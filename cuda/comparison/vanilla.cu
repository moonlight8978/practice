
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void s_scal(int n, float *alpha, float *vector) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    vector[index] = vector[index] * (*alpha);
  }
}

int main() {
  int N = 2000000;
  int threads_per_block = 256;
  int blocks_per_grid = (N + 255) / threads_per_block;

  float alpha, *vector, *d_alpha, *d_vector;

  alpha = 12.0;
  vector = new float[N];

  for (int i = 0; i < N; i += 1) {
    vector[i] = 1.0f;
  }

  hipMalloc(&d_alpha, sizeof(float));
  hipMalloc(&d_vector, N * sizeof(float));

  hipMemcpy(d_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vector, vector, N * sizeof(float), hipMemcpyHostToDevice);

  s_scal<<<blocks_per_grid, threads_per_block>>>(N, d_alpha, d_vector);

  hipMemcpy(vector, d_vector, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(vector[i]-alpha));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(d_alpha);
  hipFree(d_vector);
  free(vector);

  hipDeviceReset();

  return 0;
}
