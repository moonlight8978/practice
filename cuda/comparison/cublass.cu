#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 1000
#define N 2000
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify(
  hipblasHandle_t handle,
  float *vector,
  int ldm,
  float alpha
) {

  int start = IDX2C(0, 0, ldm);
  hipblasSscal(handle, M * N, &alpha, &vector[start], 1);
}

int main (void) {
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  int i, j;
  float *d_vector, *vector;
  float alpha = 12.0f;

  vector = new float[M * N];

  for (j = 0; j < N; j += 1) {
    for (i = 0; i < M; i += 1) {
      vector[IDX2C(i, j, M)] = 1.0f;
    }
  }

  cudaStat = hipMalloc(&d_vector, M * N * sizeof(float));
  if (cudaStat != hipSuccess) {
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
  }

  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }

  stat = hipblasSetMatrix(M, N, sizeof(float), vector, M, d_vector, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    hipFree(d_vector);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  modify(handle, d_vector, M, alpha);
  stat = hipblasGetMatrix(M, N, sizeof(float), d_vector, M, vector, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data upload failed");
    hipFree (d_vector);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }
  hipFree(d_vector);
  hipblasDestroy(handle);

  free(vector);
  hipDeviceReset();
  return EXIT_SUCCESS;
}
