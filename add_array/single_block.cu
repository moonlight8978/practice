#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <time.h>

#include <utils.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void add(int n, float* x, float* y) {
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (int i = index; i < n; i += stride) {
    y[i] += x[i];
  }
}

int main() {
  timer timer;
  int N = 1000000;
  int threads_per_block = 256;
  int blocks_per_grid = 1;
  float *x, *y, *d_x, *d_y;
  
  x = new float[N];
  y = new float[N];
  
  hipMalloc(&d_x, N * sizeof(float)); 
  hipMalloc(&d_y, N * sizeof(float));
  
  for (int i = 0; i < N; i += 1) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  } 
  
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  
  timer_start(timer);
  add<<<blocks_per_grid, threads_per_block>>>(N, d_x, d_y);
  hipDeviceSynchronize();
  timer_finish(timer);
  timer_print(timer);
  
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  hipDeviceReset();
  
  return 0;
}